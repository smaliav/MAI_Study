
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)												\
do {															\
	hipError_t res = call;										\
	if (res != hipSuccess) {									\
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n",	\
				__FILE__, __LINE__, hipGetErrorString(res));	\
		exit(0);												\
	}															\
} while (0)

__global__ void kernel(double *devVec, double *revVec, int n) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x + gridDim.x;

	while (idx < n) {
		revVec[idx] = devVec[n - 1 - idx];

		idx += offset;
	}
}

void printVector(double *vec, int size) {
	for (int i = 0; i < size; ++i) {
		printf("%f ", vec[i]);
	}
	printf("\n");
}

int main() {
	int n;			// Vector size
	double *vec;
	int blocks = 256, threads = 256;

	// Set up initial data
	scanf("%d", &n);
	vec = (double *)malloc(sizeof(double) * n);

	for (int i = 0; i < n; ++i) {
		scanf("%lf", &vec[i]);
	}

	// Create CUDA vectors and copy data to first
	double *devVec, *revVec;

	CSC(hipMalloc(&devVec, sizeof(double) * n));
	CSC(hipMalloc(&revVec, sizeof(double) * n));
	CSC(hipMemcpy(devVec, vec, sizeof(double) * n, hipMemcpyHostToDevice));

	// Call kernel
	kernel<<<blocks, threads>>>(devVec, revVec, n);
	CSC(hipGetLastError());

	// Copy results from device to host memory
	CSC(hipMemcpy(vec, revVec, sizeof(double) * n, hipMemcpyDeviceToHost));
	CSC(hipFree(devVec));
	CSC(hipFree(revVec));

	// Print results
	for (int i = 0; i < n; ++i) {
		printf("%.10e ", vec[i]);
	}

	free(vec);

	return 0;
}
