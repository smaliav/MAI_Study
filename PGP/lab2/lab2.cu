#include "hip/hip_runtime.h"
#include <stdio.h>

#define CSC(call) \
do { \
	hipError_t res = call; \
	if (res != hipSuccess) { \
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

typedef struct image {
    int w;
    int h;
    uchar4* data;
} image;

image* readImage(const char* filename) {
    image* img = (image *)malloc(sizeof(image));
    
    FILE* input = fopen(filename, "rb");

    fread(&img->w, sizeof(img->w), 1, input);
    fread(&img->h, sizeof(img->h), 1, input);
    
    img->data = (uchar4 *)malloc(sizeof(uchar4) * img->h * img->w);
    fread(&img->data, sizeof(img->data), img->h * img->w, input);

    fclose(input);

    return img;
}

void writeImage(char* filename, image* img) {
    FILE* output = fopen(filename, "wb");

    fwrite(&img->w, sizeof(img->w), 1, output);
    fwrite(&img->h, sizeof(img->h), 1, output);
    fwrite(&img->data, sizeof(img->data), img->h * img->w, output);

    fclose(output);
}

void deleteImage(image* img) {
    free(img->data);
    free(img);
    img = NULL;
}

__global__ void gaussianBlurKernel(uchar4* data,
                                   int r,
                                   float div,
                                   int w,
                                   int h,
                                   int axisX,
                                   int axisY) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    int i, j, k;
    uchar4 pxl;

    for (i = idx; i < w, i += offsetX) {
        for (j = idy; j < h, j += offsetY) {
            float r = 0.0,
                  g = 0.0,
                  b = 0.0;

            float weight = 0.0;

            for (k = -r; k <= r; k++) {
                weight = exp((float)-(k * k) / (float)(2 * r * r));
                
                int posX = pos(i + (k * axisX), w);
                int posY = pos(j + (k * axisY), h);

                pxl = tex2D(tex, (float)posX, (float)posY);

                r += (p.x) * weight;
                g += (p.y) * weight;
                b += (p.z) * weight;
            }

            data = make_uchar4((unsigned char)(r / div),
                               (unsigned char)(g / div),
                               (unsigned char)(b / div),
                               (float)0.0);
        }
    }
}

__device__ __host__ int pos(int i, int border) {
    return max(0, min(i, border));
}

texture <uchar4, 2, hipReadModeElementType> tex;

int main(int argc, char* argv[]) {
    int r;                  // Filter radius
    char srcFilename[256];
    char resFilename[256];

    scanf("%s", srcFilename);
    scanf("%s", resFilename);
    scanf("%d", r);

    image* img = readImage(srcFilename);

    hipArray *cuArr;
    hipChannelFormatDesc channel = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&cuArr, &channel, img->w, img->h));
    CSC(hipMemcpyToArray(cuArr, 0, 0, img->data, sizeof(uchar4) * img->h * img->w, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = channel;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    dim3 gridSize(32, 32);
    dim3 blockSize(32, 32);

    hipBindTextureToArray(tex, cuArr, channel);
    uchar4* tmpData;
    hipMalloc(&tmpData, sizeof(uchar4) * img->h * img->w);

    if (r > 0) {
        float div = 0.0;
        
        for (int i = -r; i <= r; i++) {
            div += exp((float)-(i * i) / (float)(2 * r * r));
        }

        gaussianBlurKernel<<<gridSize, blockSize>>>(tmpData, r, div, img->w, img->h, 1, 0);
        CSC(hipGetLastError());

        CSC(hipDeviceSynchronize());

        CSC(hipMemcpy(img->data, tmpData, sizeof(uchar4) * img->h * img->h, hipMemcpyDeviceToHost));
        CSC(hipMemcpyToArray(cuArr, 0, 0, img->data, sizeof(uchar4) * img->h * img->w, hipMemcpyHostToDevice));

        gaussianBlurKernel<<<gridSize, blockSize>>>(tmpData, r, div, img->w, img->h, 0, 1);
        CSC(hipGetLastError());

        CSC(hipDeviceSynchronize());

        CSC(hipMemcpy(img->data, tmpData, sizeof(uchar4) * img->h * img->w, hipMemcpyDeviceToHost));
    }

    writeImage(resFilename, img);

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(cuArr));
    CSC(hipFree(tmpData));

    deleteImage(img);

    return 0;
}