#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#define CSC(call) \
do { \
	hipError_t res = call; \
	if (res != hipSuccess) { \
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

typedef struct _image {
    int width;
    int height;
    uchar4* pixels;
} image;

image* readImage(const char* filename) {
    image* img = (image *)malloc(sizeof(image));
    
    FILE* input = fopen(filename, "rb");

    fread(&img->width, sizeof(img->width), 1, input);
    fread(&img->height, sizeof(img->height), 1, input);

    img->pixels = (uchar4 *)malloc(sizeof(uchar4) * img->height * img->width);
    fread(img->pixels, sizeof(img->pixels), img->height * img->width, input);
    
    fclose(input);

    return img;
}

void writeImage(const char* filename, const image* img) {
    FILE* output = fopen(filename, "wb");

    fwrite(&img->width, sizeof(img->width), 1, output);
    fwrite(&img->height, sizeof(img->height), 1, output);
    fwrite(img->pixels, sizeof(img->pixels), img->height * img->width, output);

    fclose(output);
}

void deleteImage(image* img) {
    free(img->pixels);
    free(img);
    img = NULL;
}

texture<uchar4, 2, hipReadModeElementType> tex;

__device__ __host__ int pos(int i, int border) {
    return max(min(i, border), 0);
}

__global__ void gaussianBlurKernel(uchar4* pixels,
                                   int rad,
                                   float div,
                                   int width,
                                   int height,
                                   int axisX,
                                   int axisY) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    int i, j, k;
    uchar4 pixel;

    for (i = idx; i < width; i += offsetX) {
        for (j = idy; j < height; j += offsetY) {
            float r = 0.0,
                  g = 0.0,
                  b = 0.0;

            float weight = 0.0;

            for (k = -rad; k <= rad; k++) {
                weight = exp(-(float)(k * k) / (float)(2 * rad * rad));
                
                int posX = pos(i + (k * axisX), width);
                int posY = pos(j + (k * axisY), height);

                pixel = tex2D(tex, (float)posX, (float)posY);

                r += (pixel.x) * weight;
                g += (pixel.y) * weight;
                b += (pixel.z) * weight;
            }

            pixels[i + j * width] = make_uchar4((unsigned char)(r / div),
                               (unsigned char)(g / div),
                               (unsigned char)(b / div),
                               (float)0.0);
        }
    }
}

void printPixels(uchar4* arr, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << (float)arr[i].x << " " << (float)arr[i].y << " " << (float)arr[i].z << std::endl;
    }
}

int main(int argc, char* argv[]) {
    int r;                  // Filter radius
    char srcFilename[256];
    char resFilename[256];
    
    dim3 gridSize(32, 32);
    dim3 blockSize(32, 32);

    scanf("%s", srcFilename);
    scanf("%s", resFilename);
    scanf("%d", &r);

    image* img = readImage(srcFilename);

    hipArray *cudaArr;
    hipChannelFormatDesc channel = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&cudaArr, &channel, img->width, img->height));
    CSC(hipMemcpyToArray(cudaArr, 0, 0, img->pixels,
        sizeof(uchar4) * img->height * img->width, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = channel;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    hipBindTextureToArray(tex, cudaArr, channel);

    uchar4* tmpData;
    hipMalloc(&tmpData, sizeof(uchar4) * img->height * img->width);

    if (r > 0) {
        float div = 0.0;
        
        for (int i = -r; i <= r; i++) {
            div += exp(-(float)(i * i) / (float)(2 * r * r));
        }

        gaussianBlurKernel<<<gridSize, blockSize>>>(tmpData, r, div, img->width, img->height, 1, 0);
        CSC(hipGetLastError());

        CSC(hipDeviceSynchronize());

        CSC(hipMemcpy(img->pixels, tmpData, sizeof(uchar4) * img->height * img->width, hipMemcpyDeviceToHost));
        CSC(hipMemcpyToArray(cudaArr, 0, 0, img->pixels, sizeof(uchar4) * img->height * img->width, hipMemcpyHostToDevice));

        gaussianBlurKernel<<<gridSize, blockSize>>>(tmpData, r, div, img->width, img->height, 0, 1);
        CSC(hipGetLastError());

        CSC(hipDeviceSynchronize());

        CSC(hipMemcpy(img->pixels, tmpData, sizeof(uchar4) * img->height * img->width, hipMemcpyDeviceToHost));
        // printPixels(img->pixels, img->height * img->width);
    }

    writeImage(resFilename, img);

    CSC(hipUnbindTexture(tex));
    CSC(hipFreeArray(cudaArr));
    CSC(hipFree(tmpData));

    deleteImage(img);

    return 0;
}