
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#define CSC(call) \
do { \
	hipError_t res = call; \
	if (res != hipSuccess) { \
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

#define DIM3 3
#define R 0
#define G 1
#define B 2
#define MAX_NC 32
#define N_INF_FLOAT __int_as_float(0xff800000)

typedef struct _arr3 {
    double data[DIM3];
} arr3;

typedef struct _pxlClass {
    int size;
    int* x;
    int* y;
    arr3 avg;
} pxlClass;

typedef struct _image {
    int width;
    int height;
    uchar4* pixels;
} image;

void writeImage(const char* filename, const image* img) {
    FILE* output = fopen(filename, "wb");

    fwrite(&img->width, sizeof(img->width), 1, output);
    fwrite(&img->height, sizeof(img->height), 1, output);
    fwrite(img->pixels, sizeof(img->pixels), img->height * img->width, output);

    fclose(output);
}

void deleteImage(image* img) {
    free(img->pixels);
    free(img);
    img = NULL;
}

image* readImage(const char* filename) {
    image* img = (image *)malloc(sizeof(image));
    
    FILE* input = fopen(filename, "rb");

    fread(&img->width, sizeof(img->width), 1, input);
    fread(&img->height, sizeof(img->height), 1, input);

    img->pixels = (uchar4 *)malloc(sizeof(uchar4) * img->height * img->width);
    fread(img->pixels, sizeof(img->pixels), img->height * img->width, input);
    
    fclose(input);

    return img;
}

__device__ __host__ int pos2Dto1D(int x, int y, int width) {
    return x + y * width;
}

pxlClass* readClasses(const int nc, const image* img) {
    pxlClass* pxlC = (pxlClass *)malloc(sizeof(pxlClass) * nc);
    
    for (int i = 0; i < nc; ++i) {
        int np;

        scanf("%d", &np);

        pxlC[i].x = (int *)malloc(sizeof(int) * np);
        pxlC[i].y = (int *)malloc(sizeof(int) * np);

        memset(pxlC[i].avg.data, 0.0, sizeof(pxlC[i].avg.data));
        
        for (int j = 0; j < np; ++j) {
            scanf("%d %d", &(pxlC[i].x[j]), &(pxlC[i].y[j]));

            uchar4 pixel = img->pixels[pos2Dto1D(pxlC[i].x[j], pxlC[i].y[j], img->width)];

            pxlC[i].avg.data[R] += pixel.x;
            pxlC[i].avg.data[G] += pixel.y;
            pxlC[i].avg.data[B] += pixel.z;
        }

        pxlC[i].avg.data[R] /= (double)np;
        pxlC[i].avg.data[G] /= (double)np;
        pxlC[i].avg.data[B] /= (double)np;
    }

    return pxlC;
}

__constant__ arr3 avgC[MAX_NC];

void deleteClasses(pxlClass* pxlC) {
    free(pxlC->x);
    free(pxlC->y);
    free(pxlC);

    pxlC = NULL;
}

__device__ double calcMinDist(const uchar4* pixel, int idx) {
    double dist = 0.0;
    arr3 value;

    value.data[R] = pixel->x - avgC[idx].data[R];
    value.data[G] = pixel->y - avgC[idx].data[G];
    value.data[B] = pixel->z - avgC[idx].data[B];

    for (int i = 0; i < DIM3; ++i) {
        dist += value.data[i] * value.data[i];
    }

    return -dist;
}

__global__ void minDist(uchar4* res, int width, int height, int nc) {
    int idX = blockIdx.x * blockDim.x + threadIdx.x;
    int idY = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetX = gridDim.x * blockDim.x;
    int offsetY = gridDim.y * blockDim.y;

    for (int i = idX; i < width; i += offsetX) {
        for (int j = idY; j < height; j += offsetY) {
            int pos = pos2Dto1D(i, j, width);
            double maxDist = N_INF_FLOAT;
            int maxIdx = 0;

            for (int k = 0; k < nc; ++k) {
                double dist = calcMinDist(&res[pos], k);

                if (dist > maxDist) {
                    maxDist = dist;
                    maxIdx = k;
                }
            }

            res[pos].w = maxIdx;
        }
    }
}

void printHexPixels(const image *img) {
    for (int x = 0; x < img->width; x++) {
        for (int y = 0; y < img->height; ++y) {
            uchar4 p = img->pixels[x + y * img->width];

            printf("PXL %f %f %f %f\n", (float)p.x, (float)p.y, (float)p.z, (float)p.w);
        }
    }
}

void printArray(float *arr, int size) {
    for (int i = 0; i < size; ++i) {
        printf("DEBUG %f\n", arr[i]);
    }
}

int main(int argc, char const *argv[]) {
    char srcFilename[256];
    char resFilename[256];
    int nc;

    scanf("%s", srcFilename);
    scanf("%s", resFilename);

    image *img = readImage(srcFilename);

    scanf("%d", &nc);
    pxlClass* pxlC = readClasses(nc, img);
    uchar4 *resDist;
    CSC(hipMalloc(&resDist, sizeof(uchar4) * img->width * img->height));
    CSC(hipMemcpy(resDist, img->pixels, sizeof(uchar4) * img->width * img->height, hipMemcpyHostToDevice));
    
    for (int i = 0; i < nc; ++i) {
        CSC(hipMemcpyToSymbol(HIP_SYMBOL(avgC), &(pxlC[i].avg), sizeof(arr3), i * sizeof(arr3)));
    }

    dim3 grid(16, 16);
    dim3 block(16, 16);

    minDist<<<grid, block>>>(resDist, img->width, img->height, nc);

    CSC(hipGetLastError());

    CSC(hipMemcpy(img->pixels, resDist, sizeof(uchar4) * img->height * img->width,
        hipMemcpyDeviceToHost));
    writeImage(resFilename, img);

    deleteClasses(pxlC);
    deleteImage(img);

    return 0;
}
