#include "hip/hip_runtime.h"
#include <stdio.h>

#define CSC(call) \
do { \
	hipError_t res = call; \
	if (res != hipSuccess) { \
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

#define DIM 3
#define R 0
#define G 1
#define B 2

typedef _pxlClass {
    int size;
    int* x;
    int* y;
    double avg[DIM];
} pxlClass;

typedef struct _image {
    int width;
    int height;
    uchar4* pixels;
} image;

void writeImage(const char* filename, const image* img) {
    FILE* output = fopen(filename, "wb");

    fwrite(&img->width, sizeof(img->width), 1, output);
    fwrite(&img->height, sizeof(img->height), 1, output);
    fwrite(img->pixels, sizeof(img->pixels), img->height * img->width, output);

    fclose(output);
}

void deleteImage(image* img) {
    free(img->pixels);
    free(img);
    img = NULL;
}

image* readImage(const char* filename) {
    image* img = (image *)malloc(sizeof(image));
    
    FILE* input = fopen(filename, "rb");

    fread(&img->width, sizeof(img->width), 1, input);
    fread(&img->height, sizeof(img->height), 1, input);

    img->pixels = (uchar4 *)malloc(sizeof(uchar4) * img->height * img->width);
    fread(img->pixels, sizeof(img->pixels), img->height * img->width, input);
    
    fclose(input);

    return img;
}

__device__ __host__ int pos2Dto1D(int x, int y, int width) {
    return x + y * width;
}

pxlClass* readPxlClasses(int nc, image* img) {
    pxlClass* pxlC = (pxlClass *)malloc(sizeof(pxlClass * nc));
    
    for (int i = 0; i < nc; ++i) {
        int np;

        scanf("%d", np);

        pxlC[i].x = (int *)malloc(sizeof(int) * np);
        pxlC[i].y = (int *)malloc(sizeof(int) * np);

        memset(pxlC[i].avg, 0.0, sizeof(pxlC[i].avg));
        
        for (int j = 0; j < np; ++j) {
            scanf("%d %d", &(pxlC[i].x[j]), &(pxlC[i].y[j]));

            uchar4 pixel = img->pixels[pos2Dto1D(pxlC[i].x[j], pxlC[i].y[j], img->width)];

            pxlC[i].avg[R] += pixel.x;
            pxlC[i].avg[G] += pixel.y;
            pxlC[i].avg[B] += pixel.z;
        }

        pxlC[i].avg[R] /= (double)np;
        pxlC[i].avg[G] /= (double)np;
        pxlC[i].avg[B] /= (double)np;
    }

    return pxlC;
}

int main(int argc, char** argv) {
    char srcFilename[256];
    char resFilename[256];
    int nc;

    scanf("%s", srcFilename);
    scanf("%s", resFilename);

    image *img = readImage(srcFilename);

    scanf("%d", nc);
    pxlClass* pxlC = readClasses(nc, img);

    uchar4* res = (uchar4 *)malloc(sizeof(uchar4) * img->height * img->width);
    memcpy(res, img->pixels, sizeof(uchar4) * img->height * img->width);

    // TODO Main algorithm

    return 0;
}