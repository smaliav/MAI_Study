#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include <cmath>
#include <time.h>
#include <iostream>

#define CSC(call) \
do { \
	hipError_t res = call; \
	if (res != hipSuccess) { \
		fprintf(stderr, "ERROR: file:%s line:%d message:%s\n", \
				__FILE__, __LINE__, hipGetErrorString(res)); \
		exit(0); \
	} \
} while (0)

typedef struct _comparator {
    __device__ __host__ bool operator()(double a, double b) {
        return std::fabs(a) < std::fabs(b);
    }
} comparator;

__device__ __host__ int pos2Dto1D(int x, int y, int width) {
    return x + y * width;
}

__global__ void subtractProductKernel(double *mtx, int size, int i) {
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    for (int j = idX + i + 1; j < size; j += offsetX) {
        for (int k = idY + i + 1; k < size; k += offsetY) {
            mtx[pos2Dto1D(j, k, size)] -= mtx[pos2Dto1D(j, i, size)] * mtx[pos2Dto1D(i, k, size)];
        }
    }
}

__global__ void divideKernel(double *mtx, int size, int columnIdx) {
    int idX = columnIdx + threadIdx.x + blockIdx.x * blockDim.x + 1;
    int offsetX = blockDim.x * gridDim.x;

    while (idX < size) {
        mtx[pos2Dto1D(idX, columnIdx, size)] /= mtx[pos2Dto1D(columnIdx, columnIdx, size)];
        idX += offsetX;
    }
}

__global__ void swapKernel(double *mtx, int size, int strIdx, int swapIdx) {
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int offsetX = blockDim.x * gridDim.x;
    double swapTmp;

    for (int i = idX; i < size; i += offsetX) {
        swapTmp = mtx[pos2Dto1D(strIdx, i, size)];
        mtx[pos2Dto1D(strIdx, i, size)] = mtx[pos2Dto1D(swapIdx, i, size)];
        mtx[pos2Dto1D(swapIdx, i, size)] = swapTmp;
    }
}

void printSquareMatrix(const double *mtx, const int &size) {
    for (int x = 0; x < size; ++x) {
        for (int y = 0; y < size; ++y) {
            printf("%lf ", mtx[pos2Dto1D(x, y, size)]);
        }

        printf("\n");
    }
}

void printVector(const int *vec, const int &size) {
    for (int i = 0; i < size; ++i) {
        printf("%d ", vec[i]);
    }

    printf("\n");
}

void initPermVec(int *p, const int &size) {
    for (int i = 0; i < size; ++i) {
        p[i] = i;
    }
}

void readSquareMatrix(double *mtx, const int &size) {
    for (int x = 0; x < size; ++x) {
        for (int y = 0; y < size; ++y) {
            scanf("%lf", &mtx[pos2Dto1D(x, y, size)]);  // Check %lf -> %f
        }
    }
}

void printAnswer(const double *mtx, const int *p, const int &size) {
    std::cout.precision(11);

    // Print C matrix
    for (int x = 0; x < size; ++x) {
        for (int y = 0; y < size; ++y) {
            std::cout << std::fixed << mtx[pos2Dto1D(x, y, size)] << " ";
        }

        std::cout << std::endl;
    }
    
    // Print permutation vector
    for (int i = 0; i < size; ++i) {
        std::cout << p[i] << " ";
    }

    std::cout << std::endl;
}

int main(int argc, const char **argv) {
    // TODO Implement time calculation
    comparator compare;

    int size;           // Size of initial matrix
    scanf("%d", &size);

    // Read initial matrix
    double *mtx = (double *)malloc(sizeof(double) * size * size);
    readSquareMatrix(mtx, size);
    // printSquareMatrix(mtx, size);   // DEBUG

    // Initialize permutation vector
    int *p = (int *)malloc(sizeof(int) * size);
    initPermVec(p, size);
    // printVector(p, size);   // DEBUG

    // Initialize CUDA matrix
    double *cudaMtx;
    CSC(hipMalloc(&cudaMtx, sizeof(double) * size * size));
    CSC(hipMemcpy(cudaMtx, mtx, sizeof(double) * size * size, hipMemcpyHostToDevice));

    // Main algorithm
    thrust::device_ptr<double> ptr = thrust::device_pointer_cast(cudaMtx);

    for (int i = 0; i < size - 1; ++i) {
        // Searching for max element
        thrust::device_ptr<double> maxElem = thrust::max_element(ptr + i + i * size,
            ptr + i * size + size, compare);
        
        int maxIdx = maxElem - ptr - i * size;
        p[i] = maxIdx;

        // Swap i string and string with max element
        if (maxIdx != i) {
            swapKernel<<<32, 256>>>(cudaMtx, size, i, maxIdx);
        }

        // Divide column by max element
        divideKernel<<<32, 256>>>(cudaMtx, size, i);

        // Subtract the product of Cji and Cik from Cjk
        subtractProductKernel<<<dim3(32, 32), dim3(32, 32)>>>(cudaMtx, size, i);
    }

    CSC(hipMemcpy(mtx, cudaMtx, sizeof(double) * size * size, hipMemcpyDeviceToHost));
    // printSquareMatrix(mtx, size);   // DEBUG

    printAnswer(mtx, p, size);

    CSC(hipFree(cudaMtx));
    free(mtx);
    free(p);

    return 0;
}
